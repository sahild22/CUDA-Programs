/*
 * Created by Sahil Diwan on 01/30/2017
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello(){
	printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
}

int main(int args, char **argv){
	// Launch the kernel
	hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

	// 	Force printf()s to flush
	hipDeviceSynchronize();

	printf("That's all!!\n");

	return 0;
}
